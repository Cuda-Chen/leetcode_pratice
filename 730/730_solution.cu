#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

/* https://zxi.mytechroad.com/blog/dynamic-programming/leetcode-730-count-different-palindromic-subsequences/ */

#include "hip/hip_runtime.h"


long kMod = 1000000007;

inline void checkError(hipError_t status) {
    if(status != hipSuccess) {
        fprintf(stderr, "Cuda failure %s:%d:%s\n", __FILE__, __LINE__, hipGetErrorString(status));
    }
}

__global__ void prepareData(int *dp, int n) {
  for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
    for(int j = blockIdx.y * blockDim.y + threadIdx.y; j < n; j+= blockDim.y * gridDim.y) {
      dp[i * n + j] = 0;
    }
  }
}

__global__ void setData(int *dp, int n) {
  for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
    dp[i * n + i] = 1;
  }
}

__global__ void helperKernel(char *S, int *dp, int n, long kMod, int len) {
  for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n - len; i += blockDim.x * gridDim.x) {
      int j = i + len; // jth element is the end of current string
      if(S[i] == S[j]) { // if front and rear are the same
          dp[i * n + j] = dp[(i + 1) * n + (j - 1)] * 2;
          int left = i + 1;
          int right = j - 1;
           
          while(left <= right && S[left] != S[i]) {
              left++;
          }
          while(left <= right && S[right] != S[i]) {
              right--;
          }
           
          if(left == right) {
              dp[i * n + j] += 1;
          } else if(left > right) {
              dp[i * n + j] += 2;
          } else {
              dp[i * n + j] -= dp[(left + 1) * n + (right - 1)];
          }
       } else {
           dp[i * n + j] = dp[i * n + (j - 1)] + dp[((i + 1) * n) + j] - dp[(i + 1) * n + (j - 1)];
       }
       
       dp[i * n + j] = (dp[i * n + j] + kMod) % kMod; // perform positive modulo
   }
   //__syncthreads();
}

int countPalindromicSubsequences(char * S){
    int n = strlen(S);
    int *dp = (int *)malloc(n * n * sizeof(int));
    
    // initialize the value of each element to 0
    for(int i = 0; i < n * n; i++) {
        dp[i] = 0;
    }
    
    // one char itself is its palindrom
    for(int i = 0; i < n; i++) {
        dp[(i * n) + i] = 1;
    }

    clock_t t;
    t = clock();    
    for(int len = 1; len <= n; len++) {
        for(int i = 0; i < n - len; i++) {
            int j = i + len; // jth element is the end of current string
            if(S[i] == S[j]) { // if front and rear are the same
                dp[i * n + j] = dp[(i + 1) * n + (j - 1)] * 2;
                int left = i + 1;
                int right = j - 1;
                
                while(left <= right && S[left] != S[i]) {
                    left++;
                }
                while(left <= right && S[right] != S[i]) {
                    right--;
                }
                
                if(left == right) {
                    dp[(i * n) + j] += 1;
                } else if(left > right) {
                    dp[(i * n) + j] += 2;
                } else {
                    dp[(i * n) + j] -= dp[(left + 1) * n + (right - 1)];
                }
            } else {
                dp[(i * n) + j] = dp[(i * n) + (j - 1)] + dp[((i + 1) * n) + j] - dp[((i + 1) * n) + (j - 1)];
            }
            
            dp[(i * n) + j] = (dp[(i * n) + j] + kMod) % kMod; // perform positive modulo
        }
    }
    t = clock() - t;
    printf("execution time: %f\n", ((float)t / CLOCKS_PER_SEC));
    
    // GPU part
    int *h_dp = (int *)malloc(n * n * sizeof(int));
    int *d_dp;
    char *d_S;

    checkError(hipMalloc(&d_dp, n * n * sizeof(int)));
    checkError(hipMalloc(&d_S, n * sizeof(char)));

    checkError(hipMemcpy(d_dp, h_dp, n * n * sizeof(int), hipMemcpyHostToDevice));
    checkError(hipMemcpy(d_S, S, n * sizeof(char), hipMemcpyHostToDevice));

    const dim3 blockSize(n, n, 1);
    prepareData<<<1, blockSize>>>(d_dp, n);
    setData<<<1, n>>>(d_dp, n);

    hipEvent_t start, stop;
    float elapsedTime;
    checkError(hipEventCreate(&start));
    checkError(hipEventCreate(&stop));

    checkError(hipEventRecord(start, 0));
    for(int len = 1; len <= n; len++) {
        helperKernel<<<1, n>>>(d_S, d_dp, n, kMod, len);
    }
    checkError(hipDeviceSynchronize());
    checkError(hipEventRecord(stop, 0));
    checkError(hipEventSynchronize(stop));
    checkError(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("execution time: %f\n", (elapsedTime / 1000.0));
    checkError(hipEventDestroy(start));
    checkError(hipEventDestroy(stop));

    checkError(hipMemcpy(h_dp, d_dp, n * n * sizeof(int), hipMemcpyDeviceToHost));

    for(int i = 0; i < n * n; i++) {
        if(h_dp[i] != dp[i]) {
            puts("result of GPU is inconsistant to CPU");
            break;
        }
    }
    printf("value of h_dp[0][n - 1]: %d\n", h_dp[(0 * n) + (n - 1)]);

    hipFree(d_dp);
    hipFree(d_S);

    free(h_dp);

    return dp[(0 * n) + (n - 1)];
}

int main() {
    //char *input = "bccb";
    //char *input = "abcdabcdabcdabcdabcdabcdabcdabcddcbadcbadcbadcbadcbadcbadcbadcba";
    char *input = "babbdbccbaacdbddaabdccdbacdbcdbacdaccdcbaacadbcbddbcccbbabbababdcadacdbbcbabdbaddadaababbaacddbacaaddbcaabbdbdddcbbbacbbbcbadcdadcadcdaccdcccaacadbacbaaaadcdabbbacdcbbdcbddbabacaabaadcbaaadbddbdcbbbcaacdcdbbbcdccbcdaabbddacbdcdbdcacbbdbccbccbddcacdabdcdddcbcacbadabccabaddacaaaacaabdcdbccbbdcdccbbacacaaabcdddaaaabcdbaccddabcaabcaaacacaddccbdddabbcbcaacbdcacddbdcbddbaccadbdacbaccabccdcdadacbdaccccbbccaadabacdbcbdcbadddcbcbcdbdbcdabcaacdcbbdbbbbaddbcdaaacabacaddaaccccbadbacbcbdcdacbbdbaaccdcddbcbdbbbbcbbbaaadbdbdbcbdabaacccccbddbbcccaadcbdcaacacccbcdddcbcbdacbbccdbaadddaacccbcbdadacdcdcacdccabbbdaabacdadccdadbdbcbbdcbcabdcbdccbbadbddbbbbddadbabdccbbdcbacabbbcabbcdbcabdbdbabcbddaaacacadcbcbadbdabbbddcbcbdcaacabcdbccddacbcaccadcdccaaaababccaaacbcaaaccdcaacdabddbbcbbbcccaccdaccdcbabbbdddccbcadddaaabdabacddacdbbaacbdbacaaacaaabbbcaaccddccddacabadbddcddadbbccadcdcaaccddbdabbdbddacabaacccdbdbdaccabbbcadbccccdaabbcbaacacccdcbcbabaadcbacaacbbcbccbdcdacdacddcbccdcaccaabcdbacbbdcbadabcccadadddbcaca";
    printf("input string length: %d\n", strlen(input));
    //clock_t t;
    //t = clock();
    int result = countPalindromicSubsequences(input);
    //t = clock() - t;
    //printf("execution time: %f\n", ((float)t / CLOCKS_PER_SEC));
    printf("Output: %d\n", result);
    return 0;
}
