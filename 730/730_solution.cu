#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

/* https://zxi.mytechroad.com/blog/dynamic-programming/leetcode-730-count-different-palindromic-subsequences/ */

#include "hip/hip_runtime.h"


long kMod = 1000000007;

inline void checkError(hipError_t status) {
    if(status != hipSuccess) {
        fprintf(stderr, "Cuda failure %s:%d:%s\n", __FILE__, __LINE__, hipGetErrorString(status));
    }
}

__global__ void helperKernel(char *S, int *dp, int n, long kMod) {
   for(int len = (blockIdx.x * blockDim.x + threadIdx.x) + 1; len <= n; len += blockDim.x * gridDim.x) {
       for(int i = blockIdx.y * blockDim.y + threadIdx.y; i < n - len; i += blockDim.y * gridDim.y) {
           int j = i + len; // jth element is the end of current string
           if(S[i] == S[j]) { // if front and rear are the same
               dp[i * n + j] = dp[(i + 1) * n + (j - 1)] * 2;
               int left = i + 1;
               int right = j - 1;
                
               while(left <= right && S[left] != S[i]) {
                   left++;
               }
               while(left <= right && S[right] != S[i]) {
                   right--;
               }
                
               if(left == right) {
                   dp[(i * n) + j] += 1;
               } else if(left > right) {
                   dp[(i * n) + j] += 2;
               } else {
                   dp[(i * n) + j] -= dp[(left + 1) * n + (right - 1)];
               }
            } else {
                dp[(i * n) + j] = dp[(i * n) + (j - 1)] + dp[((i + 1) * n) + j] - dp[((i + 1) * n) + (j - 1)];
            }
            
            dp[(i * n) + j] = (dp[(i * n) + j] + kMod) % kMod; // perform positive modulo
        }
    }
}

int countPalindromicSubsequences(char * S){
    int n = strlen(S);
    int *dp = (int *)malloc(n * n * sizeof(int));
    
    // initialize the value of each element to 0
    for(int i = 0; i < n * n; i++) {
        dp[i] = 0;
    }
    
    // one char itself is its palindrom
    for(int i = 0; i < n; i++) {
        dp[(i * n) + i] = 1;
    }
    
    for(int len = 1; len <= n; len++) {
        for(int i = 0; i < n - len; i++) {
            int j = i + len; // jth element is the end of current string
            if(S[i] == S[j]) { // if front and rear are the same
                dp[i * n + j] = dp[(i + 1) * n + (j - 1)] * 2;
                int left = i + 1;
                int right = j - 1;
                
                while(left <= right && S[left] != S[i]) {
                    left++;
                }
                while(left <= right && S[right] != S[i]) {
                    right--;
                }
                
                if(left == right) {
                    dp[(i * n) + j] += 1;
                } else if(left > right) {
                    dp[(i * n) + j] += 2;
                } else {
                    dp[(i * n) + j] -= dp[(left + 1) * n + (right - 1)];
                }
            } else {
                dp[(i * n) + j] = dp[(i * n) + (j - 1)] + dp[((i + 1) * n) + j] - dp[((i + 1) * n) + (j - 1)];
            }
            
            dp[(i * n) + j] = (dp[(i * n) + j] + kMod) % kMod; // perform positive modulo
        }
    }
    
    // GPU part
    int *h_dp = (int *)malloc(n * n * sizeof(int));
    int *d_dp;
    char *d_S;
    const dim3 blockSize(1, 1);
    const dim3 gridSize(n / blockSize.x + 1, n / blockSize.y + 1);

    // initialize the value of each element to 0
    for(int i = 0; i < n * n; i++) {
        h_dp[i] = 0;
    }
    
    // one char itself is its palindrom
    for(int i = 0; i < n; i++) {
        h_dp[(i * n) + i] = 1;
    }

    checkError(hipMalloc(&d_dp, n * n * sizeof(int)));
    checkError(hipMalloc(&d_S, n * sizeof(char)));

    checkError(hipMemcpy(d_dp, h_dp, n * n * sizeof(int), hipMemcpyHostToDevice));
    checkError(hipMemcpy(d_S, S, n * sizeof(char), hipMemcpyHostToDevice));

    helperKernel<<<1, 1>>>(d_S, d_dp, n, kMod);

    checkError(hipMemcpy(h_dp, d_dp, n * n * sizeof(int), hipMemcpyDeviceToHost));

    for(int i = 0; i < n * n; i++) {
        if(h_dp[i] != dp[i]) {
            puts("result of GPU is inconsistant to CPU");
            break;
        }
    }
    printf("value of dp[0][n - 1]: %d\n", h_dp[(0 * n) + (n - 1)]);

    hipFree(d_dp);
    hipFree(d_S);

    free(h_dp);

    return dp[(0 * n) + (n - 1)];
}

int main() {
    //char *input = "bccb";
    char *input = "abcdabcdabcdabcdabcdabcdabcdabcddcbadcbadcbadcbadcbadcbadcbadcba";
    clock_t t;
    t = clock();
    int result = countPalindromicSubsequences(input);
    t = clock() - t;
    printf("execution time: %f\n", ((float)t / CLOCKS_PER_SEC));
    printf("Output: %d\n", result);
    return 0;
}
